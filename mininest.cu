#include "hip/hip_runtime.h"
// starting snippet for nested sampling
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <float.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d -- %s\n",__FILE__,__LINE__, hipGetErrorString(x)); \
return EXIT_FAILURE;}} while(0)
    
#define N 32
    
__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init ( seed, idx, 0, &state[idx] );
} 
    
__global__ void generate( hiprandState* globalState, float * randomArray ) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = globalState[idx];
    float RANDOM = hiprand_uniform( &localState );
    randomArray[idx] = RANDOM;
    globalState[idx] = localState;
}


int main( int argc, char** argv) 
{
    
    dim3 threads = dim3(N, 1);
    int blocksCount = 1;// floor(N / threads.x) + 1;
    dim3 blocks  = dim3(blocksCount, 1);
    hiprandState* devStates;
    float * randomValues = new float[N];
    float * devRandomValues;
    
    printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
    blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);

    CUDA_CALL(hipMalloc ( &devStates, N*sizeof( hiprandState ) ));
    CUDA_CALL(hipMalloc ( &devRandomValues, N*sizeof( *randomValues ) ));
    
    // setup seeds
    setup_kernel <<<blocks, threads>>> ( devStates, time(NULL) );

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // generate random numbers
    generate <<<blocks, threads>>> ( devStates, devRandomValues );

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    CUDA_CALL(hipMemcpy      ( randomValues, devRandomValues, N*sizeof(*randomValues), hipMemcpyDeviceToHost ));
    
    for(int i=0;i<N;i++)
    {
        printf("#%i %f\n",i, randomValues[i]);
    }
    
    
    CUDA_CALL(hipFree(devRandomValues));
    CUDA_CALL(hipFree(devStates));
    
    delete randomValues;

    return 0;
}